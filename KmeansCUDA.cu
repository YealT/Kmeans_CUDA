#include "hip/hip_runtime.h"
#include "KmeansCUDA.h"

#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>

#include <iostream>

#include <stdlib.h>

#define BLOCKSIZE_16 16
#define BLOCKSIZE_32 32
#define OBJLENGTH 75

/**
* ���ܣ���ʼ��ÿ��������������
* �����objClusterIdx_Dev ÿ���������������
* ���룺objNum ��������
* ���룺maxIdx ���������ֵ
*/
__global__ void KmeansCUDA_Init_ObjClusterIdx(int *objClusterIdx_Dev, int objNum, int maxIdx)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x; 

	hiprandState s;
	hiprand_init(index, 0, 0, &s);

	if (index < objNum) objClusterIdx_Dev[index] = (int(hiprand_uniform(&s) * maxIdx));
}


/**
* ���ܣ����� Kmeans �ľ�������
* ���룺objData_Dev ��������
* ���룺objClusterIdx_Dev ÿ���������������
* �����clusterData_Dev ��������
* ���룺myPatameter �������
*/
__global__ void KmeansCUDA_Update_Cluster(float *objData_Dev, int *objClusterIdx_Dev, float *clusterData_Dev, sParameter myParameter)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������
	
	if (x_id < myParameter.objLength && y_id < myParameter.objNum)
	{
		int index = y_id * myParameter.objLength + x_id;
		int clusterIdx = objClusterIdx_Dev[y_id];

		atomicAdd(&clusterData_Dev[clusterIdx * myParameter.objLength + x_id], objData_Dev[index]);
	}
}

/**
*���ܣ����� Kmeans �ľ�������
* ���룺objClusterIdx_Dev ÿ���������������
* �����objNumInCluster ÿ�������е�������
* ���룺myPatameter �������
*/
__global__ void KmeansCUDA_Count_objNumInCluster(int *objClusterIdx_Dev, int *objNumInCluster, sParameter myParameter)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;

	if (index < myParameter.objNum)
	{
		int clusterIdx = objClusterIdx_Dev[index];

		atomicAdd((int*)&objNumInCluster[clusterIdx], 1); // ����
	}
}

/**
*���ܣ����� Kmeans �ľ�������
* ���룺objClusterIdx_Dev ÿ���������������
* �����objNumInCluster ÿ�������е�������
* ���룺myPatameter �������
*/
__global__ void KmeansCUDA_Count_objNumInCluster1(int *objClusterIdx_Dev, int *objNumInCluster, sParameter myParameter)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;

	__shared__ int sData[80];

	if (threadIdx.x < myParameter.clusterNum)
		sData[threadIdx.x] = 0;

	__syncthreads();

	if (index < myParameter.objNum)
	{
		int clusterIdx = objClusterIdx_Dev[index];
		atomicAdd((int*)&sData[clusterIdx], 1);
	}

	__syncthreads();

	if (threadIdx.x < myParameter.clusterNum)
		atomicAdd((int*)&objNumInCluster[threadIdx.x], sData[threadIdx.x]); // ����
}

/**
*���ܣ�ƽ�� Kmeans �ľ�������
* �����clusterData_Dev ��������
* �����objNumInCluster ÿ�������е�������
* ���룺myPatameter �������
*/
__global__ void KmeansCUDA_Scale_Cluster(float *clusterData_Dev, int *objNumInCluster, sParameter myParameter)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������
	
	if (x_id < myParameter.objLength && y_id < myParameter.clusterNum)
	{
		int index = y_id * myParameter.objLength + x_id;
		clusterData_Dev[index] /= float(objNumInCluster[y_id]);
	}
}


/**
* ���ܣ���������������ŷ������
* ���룺objects ��������
* �����clusters ������������
* ���룺objLength ��������
*/
__device__ inline static float EuclidDistance(float *objects, float *clusters, int objLength)
{
	float dist = 0.0f;

	for (int i = 0; i < objLength; i++)
	{
		float onePoint = objects[i] - clusters[i];
		dist = onePoint * onePoint + dist;
	}

	return(dist);
}

/**
* ���ܣ���������������������ĵ�ŷʽ����
* ���룺objData_Dev ��������
* ���룺objClusterIdx_Dev ÿ���������������
* ���룺clusterData_Dev ��������
* �����distOfObjAndCluster_Dev ÿ��������������ĵ�ŷʽ����
* ���룺objNumInCluster_Dev ÿ�������е�������
* ���룺iter ��������
* ���룺myPatameter �������
*/
__global__ void KmeansCUDA_distOfObjAndCluster(float *objData_Dev, int *objClusterIdx_Dev, float *clusterData_Dev, float *distOfObjAndCluster_Dev, int *objNumInCluster_Dev, int iter, sParameter myParameter)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	const int oneBlockData = OBJLENGTH * BLOCKSIZE_16;
	__shared__ float objShared[oneBlockData]; // ������
	__shared__ float cluShared[oneBlockData]; // ���������

	/* ���ݶ��빲���ڴ� */
	if (y_id < myParameter.objNum)
	{
		float *objects = &objData_Dev[myParameter.objLength * blockDim.y * blockIdx.y]; // ��ǰ����Ҫ������Ӧ���׵�ַ
		float *clusters = &clusterData_Dev[myParameter.objLength * blockDim.x * blockIdx.x]; // ��ǰ����Ҫ�������Ķ�Ӧ���׵�ַ

		for (int index = BLOCKSIZE_16 * threadIdx.y + threadIdx.x; index < oneBlockData; index = BLOCKSIZE_16 * BLOCKSIZE_16 + index)
		{
			objShared[index] = objects[index];
			cluShared[index] = clusters[index];
		}

		__syncthreads();
	}

	if (x_id < myParameter.clusterNum && y_id < myParameter.objNum)
	{
		 //if (objNumInCluster_Dev[x_id] < myParameter.minObjInClusterNum && iter >= myParameter.maxKmeansIter - 2)
			// distOfObjAndCluster_Dev[y_id * myParameter.clusterNum + x_id] = 3e30;
		 //else
			 distOfObjAndCluster_Dev[y_id * myParameter.clusterNum + x_id] = EuclidDistance(&objShared[myParameter.objLength * threadIdx.y], &cluShared[myParameter.objLength * threadIdx.x], myParameter.objLength);
	}
}

/**
* ���ܣ���������������������ĵ�ŷʽ����
* ���룺objData_Dev ��������
* ���룺objClusterIdx_Dev ÿ���������������
* ���룺clusterData_Dev ��������
* �����distOfObjAndCluster_Dev ÿ��������������ĵ�ŷʽ����
* ���룺objNumInCluster_Dev ÿ�������е�������
* ���룺iter ��������
* ���룺myPatameter �������
*/
__global__ void KmeansCUDA_distOfObjAndCluster1(float *objData_Dev, int *objClusterIdx_Dev, float *clusterData_Dev, float *distOfObjAndCluster_Dev, int *objNumInCluster_Dev, int iter, sParameter myParameter)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	__shared__ float objShared[BLOCKSIZE_16][OBJLENGTH]; // ������
	__shared__ float cluShared[BLOCKSIZE_16][OBJLENGTH]; // ���������

	float *objects = &objData_Dev[myParameter.objLength * blockDim.y * blockIdx.y]; // ��ǰ����Ҫ������Ӧ���׵�ַ
	float *clusters = &clusterData_Dev[myParameter.objLength * blockDim.x * blockIdx.x]; // ��ǰ����Ҫ�������Ķ�Ӧ���׵�ַ

	/* ���ݶ��빲���ڴ� */
	if (y_id < myParameter.objNum)
	{
		for (int xidx = threadIdx.x; xidx < OBJLENGTH; xidx += BLOCKSIZE_16)
		{
			int index = myParameter.objLength * threadIdx.y + xidx;
			objShared[threadIdx.y][xidx] = objects[index];
			cluShared[threadIdx.y][xidx] = clusters[index];
		}

		__syncthreads();
	}

	if (x_id < myParameter.clusterNum && y_id < myParameter.objNum)
	{
		if (objNumInCluster_Dev[x_id] < myParameter.minObjInClusterNum && iter >= myParameter.maxKmeansIter - 2)
			distOfObjAndCluster_Dev[y_id * myParameter.clusterNum + x_id] = 3e30;
		else
			distOfObjAndCluster_Dev[y_id * myParameter.clusterNum + x_id] = EuclidDistance(objShared[threadIdx.y], cluShared[threadIdx.x], myParameter.objLength);
	}
}

/**
* ���ܣ���������������������ĵ�ŷʽ����
* �����objClusterIdx_Dev ÿ���������������
* ���룺distOfObjAndCluster_Dev ÿ��������������ĵ�ŷʽ����
* ���룺myPatameter �������
*/
__global__ void KmeansCUDA_Update_ObjClusterIdx1(int *objClusterIdx_Dev, float *distOfObjAndCluster_Dev, sParameter myParameter)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;

	if (index < myParameter.objNum)
	{
		float *objIndex = &distOfObjAndCluster_Dev[index * myParameter.clusterNum];
		int idx = 0;
		float dist = objIndex[0];

		for (int i = 1; i < myParameter.clusterNum; i++)
		{
			if (dist > objIndex[i])
			{
				dist = objIndex[i];
				idx = i;
			}
		}
		objClusterIdx_Dev[index] = idx;
	}
}

/**
* ���ܣ���������������������ĵ�ŷʽ���루�Ż���ģ�
* �����objClusterIdx_Dev ÿ���������������
* ���룺distOfObjAndCluster_Dev ÿ��������������ĵ�ŷʽ����
* ���룺myPatameter �������
*/
__global__ void KmeansCUDA_Update_ObjClusterIdx(int *objClusterIdx_Dev, float *distOfObjAndCluster_Dev, sParameter myParameter)
{
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	__shared__ float sData[BLOCKSIZE_16][BLOCKSIZE_16]; // ������������ľ���
	__shared__ int sIndx[BLOCKSIZE_16][BLOCKSIZE_16]; // �����Ӧ������

	sData[threadIdx.y][threadIdx.x] = 2e30;
	sIndx[threadIdx.y][threadIdx.x] = 0;

	__syncthreads();

	if (y_id < myParameter.objNum)
	{
		float *objIndex = &distOfObjAndCluster_Dev[y_id * myParameter.clusterNum];
		sData[threadIdx.y][threadIdx.x] = objIndex[threadIdx.x];
		sIndx[threadIdx.y][threadIdx.x] = threadIdx.x;

		__syncthreads();

		/* ÿ BLOCKSIZE_16 �����бȽ� */
		for (int index = threadIdx.x + BLOCKSIZE_16; index < myParameter.clusterNum; index += BLOCKSIZE_16)
		{
			float nextData = objIndex[index];
			if (sData[threadIdx.y][threadIdx.x] > nextData)
			{
				sData[threadIdx.y][threadIdx.x] = nextData;
				sIndx[threadIdx.y][threadIdx.x] = index;
			}
		}

		/* BLOCKSIZE_16 �����ڲ���Լ����ֻʣ 2 �� */
		for (int step = BLOCKSIZE_16 / 2; step > 1; step = step >> 1)
		{
			int idxStep = threadIdx.x + step;
			if (threadIdx.x < step && sData[threadIdx.y][threadIdx.x] > sData[threadIdx.y][idxStep])
			{
				sData[threadIdx.y][threadIdx.x] = sData[threadIdx.y][idxStep];
				sIndx[threadIdx.y][threadIdx.x] = sIndx[threadIdx.y][idxStep];
			}
			//__syncthreads();
		}

		if (threadIdx.x == 0)
		{
			objClusterIdx_Dev[y_id] = sData[threadIdx.y][0] < sData[threadIdx.y][1] ? sIndx[threadIdx.y][0] : sIndx[threadIdx.y][1];
		}
	}
}


/**
* ���ܣ����� Kmeans ����
* ���룺objData_Host ��������
* �����objClassIdx_Host ÿ���������������
* �����centerData_Host ��������
* ���룺myPatameter �������
*/
void KmeansCUDA(float *objData_Host, int *objClassIdx_Host, float*centerData_Host, sParameter myParameter)
{
	/* �����豸���ڴ� */
	float *objData_Dev, *centerData_Dev;
	hipMalloc((void**)&objData_Dev, myParameter.objNum * myParameter.objLength * sizeof(float));
	hipMalloc((void**)&centerData_Dev, myParameter.clusterNum * myParameter.objLength * sizeof(float));
	hipMemcpy(objData_Dev, objData_Host, myParameter.objNum * myParameter.objLength * sizeof(float), hipMemcpyHostToDevice);

	int *objClassIdx_Dev;
	hipMalloc((void**)&objClassIdx_Dev, myParameter.objNum * sizeof(int));

	float *distOfObjAndCluster_Dev; // ÿ��������������ĵ�ŷʽ����
	hipMalloc((void**)&distOfObjAndCluster_Dev, myParameter.objNum * myParameter.clusterNum * sizeof(float));

	int *objNumInCluster_Dev; // ÿ�������е�������
	hipMalloc((void**)&objNumInCluster_Dev, myParameter.clusterNum * sizeof(int));


	/* �߳̿���̸߳� */
	dim3 dimBlock1D_16(BLOCKSIZE_16 * BLOCKSIZE_16);
	dim3 dimBlock1D_32(BLOCKSIZE_32 * BLOCKSIZE_32);
	dim3 dimGrid1D_16((myParameter.objNum + BLOCKSIZE_16 * BLOCKSIZE_16 - 1) / dimBlock1D_16.x);
	dim3 dimGrid1D_32((myParameter.objNum + BLOCKSIZE_32 * BLOCKSIZE_32 - 1) / dimBlock1D_32.x);

	dim3 dimBlock2D(BLOCKSIZE_16, BLOCKSIZE_16);
	dim3 dimGrid2D_Cluster((myParameter.objLength + BLOCKSIZE_16 - 1) / dimBlock2D.x, (myParameter.clusterNum + BLOCKSIZE_16 - 1) / dimBlock2D.y);
	dim3 dimGrid2D_ObjNum_Objlen((myParameter.objLength + BLOCKSIZE_16 - 1) / dimBlock2D.x, (myParameter.objNum + BLOCKSIZE_16 - 1) / dimBlock2D.y);
	dim3 dimGrid2D_ObjCluster((myParameter.clusterNum + BLOCKSIZE_16 - 1) / dimBlock2D.x, (myParameter.objNum + BLOCKSIZE_16 - 1) / dimBlock2D.y);
	dim3 dimGrid2D_ObjNum_BLOCKSIZE_16(1, (myParameter.objNum + BLOCKSIZE_16 - 1) / dimBlock2D.y);

	// ��¼ʱ��
	hipEvent_t start_GPU, end_GPU;
	float elaspsedTime;
	hipEventCreate(&start_GPU);
	hipEventCreate(&end_GPU);
	hipEventRecord(start_GPU, 0);

	/* �������������ĳ�ʼ��*/
	KmeansCUDA_Init_ObjClusterIdx<<<dimGrid1D_16, dimBlock1D_16>>>(objClassIdx_Dev, myParameter.objNum, myParameter.clusterNum);

	for (int i = 0; i < myParameter.maxKmeansIter; i++)
	{
		hipMemset(centerData_Dev, 0, myParameter.clusterNum * myParameter.objLength * sizeof(float));
		hipMemset(objNumInCluster_Dev, 0, myParameter.clusterNum * sizeof(int));

		/* ͳ��ÿһ��������� */
		KmeansCUDA_Update_Cluster<<<dimGrid2D_ObjNum_Objlen, dimBlock2D>>>(objData_Dev, objClassIdx_Dev, centerData_Dev, myParameter);

		/* ͳ��ÿһ����������� */
		//KmeansCUDA_Count_objNumInCluster1<<<dimGrid1D_16, dimBlock1D_16>>>(objClassIdx_Dev, objNumInCluster_Dev, myParameter);
		KmeansCUDA_Count_objNumInCluster<<<dimGrid1D_32, dimBlock1D_32>>>(objClassIdx_Dev, objNumInCluster_Dev, myParameter);

		/* ��������ƽ�� = ������ / �������� */
		KmeansCUDA_Scale_Cluster<<<dimGrid2D_Cluster, dimBlock2D>>>(centerData_Dev, objNumInCluster_Dev, myParameter);

		/* ����ÿ��������ÿ���������ĵ�ŷʽ���� */
		KmeansCUDA_distOfObjAndCluster<<<dimGrid2D_ObjCluster, dimBlock2D>>>(objData_Dev, objClassIdx_Dev, centerData_Dev, distOfObjAndCluster_Dev, objNumInCluster_Dev, i, myParameter);

		/* ����ÿ��������������ĵ�ŷʽ����������������ǩ */
		//KmeansCUDA_Update_ObjClusterIdx1<<<dimGrid1D_16, dimBlock1D_16>>>(objClassIdx_Dev, distOfObjAndCluster_Dev, myParameter);
		KmeansCUDA_Update_ObjClusterIdx<<<dimGrid2D_ObjNum_BLOCKSIZE_16, dimBlock2D>>>(objClassIdx_Dev, distOfObjAndCluster_Dev, myParameter);
	}

	
	// ��ʱ����
	hipEventRecord(end_GPU, 0);
	hipEventSynchronize(end_GPU);
	hipEventElapsedTime(&elaspsedTime, start_GPU, end_GPU);

	std::cout << "Kmeans ������ʱ��Ϊ��" << elaspsedTime << "ms." << std::endl;

	/* ������豸�˿������ڴ� */
	hipMemcpy(objClassIdx_Host, objClassIdx_Dev, myParameter.objNum * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(centerData_Host, centerData_Dev, myParameter.objNum * myParameter.objLength * sizeof(float), hipMemcpyDeviceToHost);

	/* �ͷ��豸���ڴ� */
	hipFree(objData_Dev);
	hipFree(objClassIdx_Dev);
	hipFree(centerData_Dev);
	hipFree(distOfObjAndCluster_Dev);
	hipFree(objNumInCluster_Dev);
}