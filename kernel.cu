#include "KmeansCUDA.h"

#include <hip/hip_runtime.h>
#include <>

#include "ClassParameter.h"
#include "ReadSaveImage.h"

#include <iostream>

using std::cout;
using std::endl;

int main()
{
	sParameter myParameter{63504, 75, 80, 40, 150, 14};

	float *objData = (float*)malloc(myParameter.objNum * myParameter.objLength * sizeof(float));
	float *centerData = (float*)malloc(myParameter.clusterNum * myParameter.objLength * sizeof(float));
	int *objClassIdx = (int*)malloc(myParameter.objNum * sizeof(int));

	ReadData(objData, myParameter);

	KmeansCUDA(objData, objClassIdx, centerData, myParameter);

	SaveData(objClassIdx, myParameter);
	
	hipDeviceReset();
	return 0;
}


